#include "hip/hip_runtime.h"
#include "utils.h"
#include <time.h>
#include <sys/time.h>
#include <math.h>

#define NR_RUNS 5

int main(int argc,char** argv){
  srand(time(NULL));
  
	if(argc < 2){
		fprintf(stderr,"ERROR: .mtx input file not provided");
		return 1;
	}

	struct timeval start={0,0};
	struct timeval end={0,0};

	double total_time = 1.0;
	float kernel_time;
	char* mtx_name = argv[1];

	hipEvent_t kernel_start;
	hipEvent_t kernel_end;

	for(int run = -NR_RUNS; run < NR_RUNS; run++){ 

		//import sparse matrix and calculate elapsed time
		gettimeofday(&start,(struct timezone*)0);

		SpM input_spm = import_spm(mtx_name);
		get_csr_repr(&input_spm);	
		double* input_vec = gen_random_vec(input_spm.tot_cols);

		gettimeofday(&end,(struct timezone*)0);
		//compute csr multiplication and get elapsed time

		hipEventCreate(&kernel_start);
		hipEventCreate(&kernel_end);
		double* res = (double*)malloc(input_spm.tot_rows*sizeof(double));

		hipEventRecord(kernel_start);
		csr_mult<<<2,50>>>(input_spm,input_vec,res); 
		hipEventRecord(kernel_end);
		hipEventSynchronize(kernel_end);
		hipDeviceSynchronize();

		kernel_time = 0;
		hipEventElapsedTime(&kernel_time,kernel_start,kernel_end); 

		// update cumulated variables
		if(run >= 0)
			total_time *= kernel_time+TIME_INTERVAL(start,end)

		hipEventDestroy(kernel_start);
		hipEventDestroy(kernel_end);
		free(res);
		free(input_vec);
	}

	double total_mean = pow(total_time,1.0/NR_RUNS);

	//calculate geometric mean and display time results
	fprintf(stdout,"%s, total elapsed time: %f\n",mtx_name,total_mean);

	return 0;
}
