#include "hip/hip_runtime.h"
#include "utils.h"
#include <time.h>
#include <sys/time.h>
#include <math.h>

#define NR_RUNS 5

void alloc_dev_buffs(SpM input_spm,uint32_t** kernel_rows,uint32_t** kernel_cols,double** kernel_values){
	hipMalloc((void**)kernel_rows,(input_spm.tot_rows+1)*sizeof(uint32_t));
	hipMalloc((void**)kernel_cols,input_spm.dim*sizeof(uint32_t));
	hipMalloc((void**)kernel_values,input_spm.dim*sizeof(double));

	hipMemcpy(*kernel_rows,input_spm.row,(input_spm.tot_rows+1)*sizeof(uint32_t),hipMemcpyHostToDevice);
	hipMemcpy(*kernel_cols,input_spm.col,input_spm.dim*sizeof(uint32_t),hipMemcpyHostToDevice);
	hipMemcpy(*kernel_values,input_spm.value,input_spm.dim*sizeof(double),hipMemcpyHostToDevice);
}

int main(int argc,char** argv){
  srand(time(NULL));
  
	if(argc < 2){
		fprintf(stderr,"ERROR: .mtx input file not provided");
		return 1;
	}

	struct timeval start={0,0};
	struct timeval end={0,0};

	double total_time = 1.0;
	double host_time= 1.0;
	double device_time = 1.0;

	float kernel_time;
	char* mtx_name = argv[1];

	uint16_t nr_blocks = 512;
	uint16_t block_dim;

	uint32_t* kernel_rows;
	uint32_t* kernel_cols;
	double* kernel_values;

	hipEvent_t kernel_start;
	hipEvent_t kernel_end;
	kernel_time = 0;

	hipEvent_t r_start;
	hipEvent_t r_stop;
	float r_time = 0;

	for(int run = -NR_RUNS; run < NR_RUNS; run++){ 

		//import sparse matrix and calculate elapsed time
		gettimeofday(&start,(struct timezone*)0);

		SpM input_spm = import_spm(mtx_name);
		get_csr_repr(&input_spm);	
		double seed = (double)rand()*((double)RAND_MAX/2.0);

		gettimeofday(&end,(struct timezone*)0);

		block_dim = input_spm.tot_rows / nr_blocks;

		double* input_vec;
		hipMallocManaged(&input_vec,input_spm.tot_cols*sizeof(double));

		hipEventCreate(&r_start);
		hipEventCreate(&r_stop);
		hipEventRecord(r_start);

		gen_random_vec<<<nr_blocks,block_dim>>>(input_vec,seed,input_spm.tot_cols);

		hipEventRecord(r_stop);
		hipEventSynchronize(r_stop);
		hipEventElapsedTime(&r_time,r_start,r_stop);
		r_time *= 1e3;
		
		//since structs with dynamic allocated arrays are not allowed
		//it is necessary to copy the arrays of the sparse matrix
		alloc_dev_buffs(input_spm,&kernel_rows,&kernel_cols,&kernel_values);	

		hipEventCreate(&kernel_start);
		hipEventCreate(&kernel_end);

		double* res;
		hipMallocManaged(&res,input_spm.tot_rows*sizeof(double));


		hipEventRecord(kernel_start);
		csr_mult<<<nr_blocks,block_dim>>>(input_spm.tot_rows,kernel_rows,kernel_cols,kernel_values,input_vec,res); 
		hipEventRecord(kernel_end);
		hipEventSynchronize(kernel_end);

		hipEventElapsedTime(&kernel_time,kernel_start,kernel_end); 
		kernel_time *= 1e3;

		// update cumulated variables
		if(run >= 0){
			total_time *= r_time+kernel_time+TIME_INTERVAL(start,end)
			host_time *= TIME_INTERVAL(start,end)
			device_time *= (kernel_time+r_time);
		}

		hipEventDestroy(kernel_start);
		hipEventDestroy(kernel_end);

		hipFree(kernel_rows);
		hipFree(kernel_cols);
		hipFree(kernel_values);
		hipFree(res);
		free_spm(&input_spm);
		hipFree(input_vec);
	}

	//calculate geometric mean and display time results
	double total_mean = pow(total_time,1.0/NR_RUNS);
	double host_mean = pow(host_time,1.0/NR_RUNS);
	double device_mean = pow(device_time,1.0/NR_RUNS);

	fprintf(stdout,"%s, host time: %f, device time: %f, total time: %f\n",mtx_name,host_mean,device_mean,total_mean);

	return 0;
}
