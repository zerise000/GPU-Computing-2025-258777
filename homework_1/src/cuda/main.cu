#include "hip/hip_runtime.h"
#include "utils.h"
#include <time.h>
#include <sys/time.h>
#include <math.h>

#define NR_RUNS 5

float dev_getInput_vec(float blocks_ratio,uint32_t dim_vec,double* input_vec,double seed){
	float elapsed_time = 0;
	hipEvent_t start;
	hipEvent_t stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	uint32_t nr_blocks = dim_vec*blocks_ratio;
	uint32_t block_dim = dim_vec/nr_blocks;

	gen_random_vec<<<nr_blocks,block_dim>>>(input_vec,seed,dim_vec);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time,start,stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsed_time*1e3;
}

float dev_csr_mult(float blocks_ratio, uint32_t nr_rows,uint32_t* kernel_rows,uint32_t* kernel_cols, double* kernel_values, double* input_vec,double* res){

	hipEvent_t kernel_start;
	hipEvent_t kernel_end;
	float mult_time = 0;
	uint32_t nr_blocks = nr_rows*blocks_ratio;
	uint32_t block_dim = nr_rows/nr_blocks;

	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_end);

	hipEventRecord(kernel_start);
	csr_mult<<<nr_blocks,block_dim>>>(nr_rows,kernel_rows,kernel_cols,kernel_values,input_vec,res); 
	hipEventRecord(kernel_end);
	hipEventSynchronize(kernel_end);

	hipEventElapsedTime(&mult_time,kernel_start,kernel_end); 

	hipEventDestroy(kernel_start);
	hipEventDestroy(kernel_end);

	return mult_time * 1e3;
}

void alloc_dev_buffs(SpM input_spm,uint32_t** kernel_rows,uint32_t** kernel_cols,double** kernel_values){
	hipMalloc((void**)kernel_rows,(input_spm.tot_rows+1)*sizeof(uint32_t));
	hipMalloc((void**)kernel_cols,input_spm.dim*sizeof(uint32_t));
	hipMalloc((void**)kernel_values,input_spm.dim*sizeof(double));

	hipMemcpy(*kernel_rows,input_spm.row,(input_spm.tot_rows+1)*sizeof(uint32_t),hipMemcpyHostToDevice);
	hipMemcpy(*kernel_cols,input_spm.col,input_spm.dim*sizeof(uint32_t),hipMemcpyHostToDevice);
	hipMemcpy(*kernel_values,input_spm.value,input_spm.dim*sizeof(double),hipMemcpyHostToDevice);
}

int main(int argc,char** argv){
  srand(time(NULL));
  
	if(argc < 2){
		fprintf(stderr,"ERROR: .mtx input file not provided");
		return 1;
	}

	struct timeval start={0,0};
	struct timeval end={0,0};

	double total_time = 1.0;
	double host_time= 1.0;
	double device_time = 1.0;
	float blocks_ratio = 0.7;

	char* mtx_name = argv[1];

	uint32_t* kernel_rows;
	uint32_t* kernel_cols;
	double* kernel_values;

	double* input_vec;
	double* res;

	for(int run = -NR_RUNS; run < NR_RUNS; run++){ 

		//host code
		gettimeofday(&start,(struct timezone*)0);

		SpM input_spm = import_spm(mtx_name);
		get_csr_repr(&input_spm);	
		double seed = (double)rand()*((double)RAND_MAX/2.0);

		hipMallocManaged(&input_vec,input_spm.tot_cols*sizeof(double));
		hipMallocManaged(&res,input_spm.tot_rows*sizeof(double));
	
		//since struct arrays are stored in host memory,copy to device memory
		alloc_dev_buffs(input_spm,&kernel_rows,&kernel_cols,&kernel_values);	

		gettimeofday(&end,(struct timezone*)0);

		//exec GPU kernels and retrieve time execution
		float tmp_dev_time = dev_getInput_vec(blocks_ratio,input_spm.tot_cols,input_vec,seed);
		tmp_dev_time += dev_csr_mult(blocks_ratio,input_spm.tot_rows,kernel_rows,kernel_cols,kernel_values,input_vec,res);
	

		// update cumulated variables
		if(run >= 0){
			total_time *= tmp_dev_time+TIME_INTERVAL(start,end)
			host_time *= TIME_INTERVAL(start,end)
			device_time *= tmp_dev_time;
		}


		hipFree(kernel_rows);
		hipFree(kernel_cols);
		hipFree(kernel_values);
		hipFree(res);
		free_spm(&input_spm);
		hipFree(input_vec);
	}

	//calculate geometric mean and display time results
	double total_mean = pow(total_time,1.0/NR_RUNS);
	double host_mean = pow(host_time,1.0/NR_RUNS);
	double device_mean = pow(device_time,1.0/NR_RUNS);

	fprintf(stdout,"%s, host time: %f, device time: %f, total time: %f\n",mtx_name,host_mean,device_mean,total_mean);

	return 0;
}
